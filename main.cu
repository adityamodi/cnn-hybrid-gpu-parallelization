#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <cmath>
#include <ctime>
#include <cfloat>

#include <algorithm>
#include <chrono>
#include <iomanip>
#include <iostream>
#include <map>
#include <memory>
#include <random>
#include <sstream>
#include <string>
#include <vector>

#include <hip/hip_runtime.h>
#include <>

#include <hipblas.h>
#include <hipDNN.h>

#include "readubyte.h"

#define BLOCK_WIDTH 128
#define GPU_ID 0
#define NUM_ITERATIONS 1000
#define RANDOM_SEED 1
#define BATCH_SIZE 64
#define STEP_SIZE 0.01


int ceilfn(int x, int y)
{
    return (x + y - 1) / y;
}

struct ConvBiasLayer
{
    int in_channels, out_channels, filter_size;
    int in_width, in_height, out_width, out_height;
    std::vector<float> pconv, pbias;
    ConvBiasLayer(int in_channels_, int out_channels_, int filter_size_,
                  int width_, int height_) : pconv(in_channels_ * filter_size_ * filter_size_ * out_channels_),
                  pbias(out_channels_)
    {
        in_channels = in_channels_;
        out_channels = out_channels_;
        filter_size = filter_size_;
        in_width = width_;
        in_height = height_;
        out_width = width_ - filter_size_ + 1;
        out_height = height_ - filter_size_ + 1;
    }
};

struct MaxPoolLayer
{
    int size, int stride;
    MaxPoolLayer(int size_, int stride_) : size(size_), stride(stride_) {}
};

struct FullyConnectedLayer
{
    int inputs, outputs;
    std::vector<float> pneurons, pbias;

    FullyConnectedLayer(int inputs_, int outputs_) : inputs(inputs_), outputs(outputs_),
                        pneurons(inputs_ * outputs_), pbias(outputs_) {}
};

__global__ void FillOnes(float *vec, int size)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx >= size)
        return;
    vec[idx] = 1.0f;
}

__global__ void SoftmaxLossBackprop(float *label, int num_labels, int batch_size, float *diff)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx >= batch_size)
        return;
    int label_value = static_cast<int>(label[idx]);
    diff[idx * num_labels + label_value] -= 1.0f;
}

struct TrainingContext
{
    hipdnnHandle_t cudnnHandle;
    hipblasHandle_t cublasHandle;

    hipdnnTensorDescriptor_t dataTensor, conv1Tensor, conv1BiasTensor, pool1Tensor,
                            conv2Tensor, conv2BiasTensor, pool2Tensor, fc1Tensor, fc2Tensor;
    hipdnnFilterDescriptor_t conv1filterDesc, conv2filterDesc;
    hipdnnConvolutionFwdAlgo_t conv1algo, conv2algo;
    hipdnnConvolutionBwdFilterAlgo_t conv1bwfalgo, conv2bwfalgo;
    hipdnnConvolutionBwdDataAlgo_t conv2bwdalgo;
    hipdnnPoolingDescriptor_t poolDesc;
    hipdnnActivationDescriptor_t fc1Activation;

    int m_gpuid;
    int m_batchsize;
    size_t m_workspaceSize;

    FullyConnectedLayer& ref_fc1, ref_fc2;

    TrainingContext& operator=(const TrainingContext&) = delete;
    TrainingContext(const TrainingContext&) = delete;
    TrainingContext(int gpuid, int batch_size, ConvBiasLayer& conv1, MaxPoolLayer& pool1, ConvBiasLayer& conv2, MaxPoolLayer& pool2,
                    FullyConnectedLayer& fc1, FullyConnectedLayer& fc2) : ref_fc1(fc1), ref_fc2(fc2), m_gpuid(gpuid)
    {
        m_batchsize = batch_size;

        checkCudaErrors(hipSetDevice(gpuid));
        checkCudaErrors(hipblasCreate(&cublasHandle));
        checkCUDNN(hipdnnCreate(&cudnnHandle));
        // Create tensor descriptors
        checkCUDNN(hipdnnCreateTensorDescriptor(&dataTensor));
        checkCUDNN(hipdnnCreateTensorDescriptor(&conv1Tensor));
        checkCUDNN(hipdnnCreateTensorDescriptor(&conv1BiasTensor));
        checkCUDNN(hipdnnCreateTensorDescriptor(&pool1Tensor));
        checkCUDNN(hipdnnCreateTensorDescriptor(&conv2Tensor));
        checkCUDNN(hipdnnCreateTensorDescriptor(&conv2BiasTensor));
        checkCUDNN(hipdnnCreateTensorDescriptor(&pool2Tensor));
        checkCUDNN(hipdnnCreateTensorDescriptor(&fc1Tensor));
        checkCUDNN(hipdnnCreateTensorDescriptor(&fc2Tensor));

        checkCUDNN(hipdnnCreateActivationDescriptor(&fc1Activation));

        checkCUDNN(hipdnnCreateFilterDescriptor(&conv1filterDesc));
        checkCUDNN(hipdnnCreateFilterDescriptor(&conv2filterDesc));

        checkCUDNN(hipdnnCreateConvolutionDescriptor(&conv1Desc));
        checkCUDNN(hipdnnCreateConvolutionDescriptor(&conv2Desc));

        checkCUDNN(hipdnnCreatePoolingDescriptor(&poolDesc));


        // Set tensor descriptor sizes
        checkCUDNN(hipdnnSetTensor4dDescriptor(conv1BiasTensor,
                                              HIPDNN_TENSOR_NCHW,
                                              HIPDNN_DATA_FLOAT,
                                              1, conv1.out_channels,
                                              1, 1));
        checkCUDNN(hipdnnSetTensor4dDescriptor(conv2BiasTensor,
                                              HIPDNN_TENSOR_NCHW,
                                              HIPDNN_DATA_FLOAT,
                                              1, conv2.out_channels,
                                              1, 1));

        checkCUDNN(hipdnnSetPooling2dDescriptor(poolDesc,
                                               HIPDNN_POOLING_MAX,
                                               HIPDNN_PROPAGATE_NAN,
                                               pool1.size, pool1.size,
                                               0, 0,
                                               pool1.stride, pool1.stride));
        checkCUDNN(hipdnnSetTensor4dDescriptor(pool2Tensor,
                                              HIPDNN_TENSOR_NCHW,
                                              HIPDNN_DATA_FLOAT,
                                              batch_size, conv2.out_channels,
                                              conv2.out_height / pool2.stride,
                                              conv2.out_width / pool2.stride));

        checkCUDNN(hipdnnSetTensor4dDescriptor(fc1Tensor,
                                              HIPDNN_TENSOR_NCHW,
                                              HIPDNN_DATA_FLOAT,
                                              batch_size, fc1.outputs, 1, 1));

        checkCUDNN(hipdnnSetTensor4dDescriptor(fc2Tensor,
                                              HIPDNN_TENSOR_NCHW,
                                              HIPDNN_DATA_FLOAT,
                                              batch_size, fc2.outputs, 1, 1));

        checkCUDNN(hipdnnSetActivationDescriptor(fc1Activation, HIPDNN_ACTIVATION_RELU,
                                                HIPDNN_PROPAGATE_NAN, 0.0));


        // Set convolution tensor sizes and compute workspace size
        size_t workspace = 0;
        workspace = std::max(workspace, SetFwdConvolutionTensors(conv1, dataTensor, conv1Tensor, conv1filterDesc, conv1Desc, conv1algo));
        workspace = std::max(workspace, SetBwdConvolutionTensors(dataTensor, conv1Tensor, conv1filterDesc, conv1Desc, &conv1bwfalgo, nullptr));

        workspace = std::max(workspace, SetFwdConvolutionTensors(conv2, pool1Tensor, conv2Tensor, conv2filterDesc, conv2Desc, conv2algo));
        workspace = std::max(workspace, SetBwdConvolutionTensors(pool1Tensor, conv2Tensor, conv2filterDesc, conv2Desc, &conv2bwfalgo, &conv2bwdalgo));

        // The workspace is allocated later (if necessary)
        m_workspaceSize = workspace;
    }

    ~TrainingContext()
    {
        checkCudaErrors(hipSetDevice(m_gpuid));

        checkCudaErrors(hipblasDestroy(cublasHandle));
        checkCUDNN(hipdnnDestroy(cudnnHandle));
        checkCUDNN(hipdnnDestroyTensorDescriptor(dataTensor));
        checkCUDNN(hipdnnDestroyTensorDescriptor(conv1Tensor));
        checkCUDNN(hipdnnDestroyTensorDescriptor(conv1BiasTensor));
        checkCUDNN(hipdnnDestroyTensorDescriptor(pool1Tensor));
        checkCUDNN(hipdnnDestroyTensorDescriptor(conv2Tensor));
        checkCUDNN(hipdnnDestroyTensorDescriptor(conv2BiasTensor));
        checkCUDNN(hipdnnDestroyTensorDescriptor(pool2Tensor));
        checkCUDNN(hipdnnDestroyTensorDescriptor(fc1Tensor));
        checkCUDNN(hipdnnDestroyTensorDescriptor(fc2Tensor));
        checkCUDNN(hipdnnDestroyActivationDescriptor(fc1Activation));
        checkCUDNN(hipdnnDestroyFilterDescriptor(conv1filterDesc));
        checkCUDNN(hipdnnDestroyFilterDescriptor(conv2filterDesc));
        checkCUDNN(hipdnnDestroyConvolutionDescriptor(conv1Desc));
        checkCUDNN(hipdnnDestroyConvolutionDescriptor(conv2Desc));
        checkCUDNN(hipdnnDestroyPoolingDescriptor(poolDesc));
    }

    
};
